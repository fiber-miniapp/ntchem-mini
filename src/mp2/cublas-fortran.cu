
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctype.h>
#include <assert.h>
#include <hipblas.h>

#define ERR_EQ(X,Y) do { if ((X) == (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
    exit(-1);}} while(0)

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
    fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
    exit(-1);}} while(0)

#define CUDA_CALL(X)   ERR_NE(X,hipSuccess)
#define CUBLAS_CALL(X) ERR_NE(X,HIPBLAS_STATUS_SUCCESS)

// #define DPRINT(X, ...) fprintf( stderr, "[%s] " X, __func__, __VA_ARGS__ )
#define DPRINT(X, ...)

/*
 *
 */
#define MAX_STREAM  8
#define MAX_EVENT  (MAX_STREAM*2)

static hipblasHandle_t  handle;
static hipStream_t    stream[MAX_STREAM+1];
static hipEvent_t     event [MAX_EVENT+1];
static hipEvent_t     event2[MAX_EVENT+1];

static int  init_flag = 0;

extern "C"
void cublas_init_()
{
    if ( init_flag ) return;

    CUBLAS_CALL( hipblasCreate( &handle ) );
    for ( int i = 0; i <= MAX_STREAM; i++ ) {
	CUDA_CALL( hipStreamCreate( &stream[i] ) );
    }
    for ( int i = 0; i <= MAX_EVENT; i++ ) {
	CUDA_CALL( hipEventCreate( &event[i] ) );
	CUDA_CALL( hipEventCreate( &event2[i] ) );
    }

    init_flag = 1;
}

extern "C"
void cublas_fin_()
{
    if ( ! init_flag ) return;

    for ( int i = 0; i <= MAX_EVENT; i++ ) {
	CUDA_CALL( hipEventDestroy( event[i] ) );
	CUDA_CALL( hipEventDestroy( event2[i] ) );
    }
    for ( int i = 0; i <= MAX_STREAM; i++ ) {
	CUDA_CALL( hipStreamDestroy( stream[i] ) );
    }
    CUBLAS_CALL( hipblasDestroy( handle ) );

    init_flag = 0;
}

extern "C"
void cublas_alloc_( void **devptr, const int *i, const int *j )
{
    size_t  size = sizeof(double) * i[0] * j[0];
    DPRINT( "size:%ld (%ld x %ld x %ld)\n", size, sizeof(double), i[0], j[0] );
    void  *buf;
    CUDA_CALL( hipMalloc( &buf, size ) );
    *devptr = buf;
    DPRINT( "devptr:%p, size:%ld (%ld x %ld x %ld)\n", *devptr, size, sizeof(double), i[0], j[0] );
}

extern "C"
void cublas_free_( void **devptr )
{
    DPRINT( "devptr:%p\n", *devptr );
    void  *buf;
    buf = *devptr;
    CUDA_CALL( hipFree( buf ) );
}

extern "C"
void cublas_set_matrix_( const int *i, const int *j, const void *A, const int *lda,
		      void **devptr_B, const int *ldb )
{
    DPRINT( "A:%p, devptr_B:%p\n", A, *devptr_B );
    double  *dev_B;
    dev_B = (double*) *devptr_B;
    CUBLAS_CALL( hipblasSetMatrix( *i, *j, sizeof(double), A, *lda, dev_B, *ldb ) );
}

extern "C"
void cublas_set_matrix_async_( const int *i, const int *j, const void *A, const int *lda,
			       void **devptr_B, const int *ldb, const int *id_st )
{
    DPRINT( "A:%p, devptr_B:%p, st:%d\n", A, *devptr_B, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    double  *dev_B;
    dev_B = (double*) *devptr_B;
    CUBLAS_CALL( hipblasSetMatrixAsync( *i, *j, sizeof(double), A, *lda, dev_B, *ldb, stream[*id_st] ) );
}

extern "C"
void cublas_get_matrix_( const int *i, const int *j, const void **devptr_A, const int *lda, 
		      void *B, const int *ldb )
{
    DPRINT( "devptr_A:%p, B:%p\n", *devptr_A, B );
    double  *dev_A;
    dev_A = (double*) *devptr_A;
    CUBLAS_CALL( hipblasGetMatrix( *i, *j, sizeof(double), dev_A, *lda, B, *ldb ) );
}

extern "C"
void cublas_get_matrix_async_( const int *i, const int *j, const void **devptr_A, const int *lda, 
			       void *B, const int *ldb, const int *id_st )
{
    DPRINT( "devptr_A:%p, B:%p, st:%d\n", *devptr_A, B, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    double  *dev_A;
    dev_A = (double*) *devptr_A;
    CUBLAS_CALL( hipblasGetMatrixAsync( *i, *j, sizeof(double), dev_A, *lda, B, *ldb, stream[*id_st] ) );
}

extern "C"
void cublas_dgemm_( const char *ta, const char *tb,
		 const int *m, const int *n, const int *k,
		 const double *alpha,
		 const void **devptr_A, const int *lda,
		 const void **devptr_B, const int *ldb,
		 const double *beta,
		 void **devptr_C, const int *ldc )
{
    DPRINT( "m:%d, n:%d, k:%d\n", *m, *n, *k );
    hipblasOperation_t  transa = HIPBLAS_OP_N;
    hipblasOperation_t  transb = HIPBLAS_OP_N;
    if ( toupper(ta[0]) == 'T' ) transa = HIPBLAS_OP_T;
    if ( toupper(tb[0]) == 'T' ) transb = HIPBLAS_OP_T;
    const double  *dev_A;
    const double  *dev_B;
    double  *dev_C;
    dev_A = (double*) *devptr_A;
    dev_B = (double*) *devptr_B;
    dev_C = (double*) *devptr_C;
    CUBLAS_CALL ( hipblasDgemm( handle, transa, transb, *m, *n, *k, alpha,
			       dev_A, *lda,
			       dev_B, *ldb, beta,
			       dev_C, *ldc ) );
}

extern "C"
void cublas_dgemm_async_( const char *ta, const char *tb,
			  const int *m, const int *n, const int *k,
			  const double *alpha,
			  const void **devptr_A, const int *lda,
			  const void **devptr_B, const int *ldb,
			  const double *beta,
			  void **devptr_C, const int *ldc,
			  const int *id_st )
{
    DPRINT( "m:%d, n:%d, k:%d, st:%d\n", *m, *n, *k, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    hipblasOperation_t  transa = HIPBLAS_OP_N;
    hipblasOperation_t  transb = HIPBLAS_OP_N;
    if ( toupper(ta[0]) == 'T' ) transa = HIPBLAS_OP_T;
    if ( toupper(tb[0]) == 'T' ) transb = HIPBLAS_OP_T;
    const double  *dev_A;
    const double  *dev_B;
    double  *dev_C;
    dev_A = (double*) *devptr_A;
    dev_B = (double*) *devptr_B;
    dev_C = (double*) *devptr_C;
    CUBLAS_CALL ( hipblasSetStream( handle, stream[*id_st] ) );
    CUBLAS_CALL ( hipblasDgemm( handle, transa, transb, *m, *n, *k, alpha,
			       dev_A, *lda,
			       dev_B, *ldb, beta,
			       dev_C, *ldc ) );
    CUBLAS_CALL ( hipblasSetStream( handle, NULL ) );
}

extern "C"
void cublas_st_sync_( const int *id_st )
{
    DPRINT( "st:%d\n", *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    CUDA_CALL( hipStreamSynchronize( stream[*id_st] ) );
}

extern "C"
void cublas_ev_rec_( const int *id_ev, const int *id_st )
{
    DPRINT( "ev:%d, st:%d\n", *id_ev, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    assert( *id_ev >= 0 && *id_ev <= MAX_EVENT );
    CUDA_CALL( hipEventRecord( event[*id_ev], stream[*id_st] ) );
}

extern "C"
void cublas_ev_wait_( const int *id_ev, const int *id_st )
{
    DPRINT( "ev:%d, st:%d\n", *id_ev, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    assert( *id_ev >= 0 && *id_ev <= MAX_EVENT );
    CUDA_CALL( hipStreamWaitEvent( stream[*id_st], event[*id_ev], 0 ) );
}

extern "C"
void cublas_ev2_rec_( const int *id_ev, const int *id_st )
{
    DPRINT( "ev:%d, st:%d\n", *id_ev, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    assert( *id_ev >= 0 && *id_ev <= MAX_EVENT );
    CUDA_CALL( hipEventRecord( event2[*id_ev], stream[*id_st] ) );
}

extern "C"
void cublas_ev2_wait_( const int *id_ev, const int *id_st )
{
    DPRINT( "ev:%d, st:%d\n", *id_ev, *id_st );
    assert( *id_st >= 0 && *id_st <= MAX_STREAM );
    assert( *id_ev >= 0 && *id_ev <= MAX_EVENT );
    CUDA_CALL( hipStreamWaitEvent( stream[*id_st], event2[*id_ev], 0 ) );
}
